/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);




/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
__host__ void check_param(int *tpoints,int *nsteps)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((*tpoints < MINPOINTS) || (*tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      *tpoints = atoi(tchar);
      if ((*tpoints < MINPOINTS) || (*tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((*nsteps < 1) || (*nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      *nsteps = atoi(tchar);
      if ((*nsteps < 1) || (*nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", *tpoints, *nsteps);

}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/

__device__ float do_math(float currentvalue,float oldval)
{
   float dtime, c, dx, tau, sqtau;
  
   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   return ((2.0 * currentvalue) - oldval + (sqtau *  (-2.0)*currentvalue));

}



/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__ void init_line(float *values,int tpoint,int nsteps)
{
   float x, fac;
   float currentval,oldval,newval;
   int indx = blockIdx.x * blockDim.x + threadIdx.x+1;

   
   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   
   x = (indx -1.0)/(float)(tpoint-1.0);
   currentval = sin(fac * x);
   oldval= currentval;
   
   /*
        k = 0.0; 
        tmp = tpoints - 1;
        for (j = 1; j <= tpoints; j++) {
            x = k/tmp;
            values[j] = sin (fac * x);
            k = k + 1.0;
        } 
   */

    /* Initialize old values array */
    /*
        for (i = 1; i <= tpoints; i++) 
            oldval[i] = values[i];
    */
    /**********************************************************************
    *     Update all values along line a specified number of times
    *********************************************************************/
    int i;
    #pragma unroll 1024
    for ( i = 1; i<=nsteps; i++) {
        if (indx ==0 || indx == tpoint)
            currentval =0.0;
        else{
            newval =  do_math(currentval,oldval);
            oldval = currentval;
            currentval = newval;
        }
    }
    values[indx] = currentval;


}





/**********************************************************************
 *     Print final results
 *********************************************************************/
__host__ void printfinal(float values[],int tpoints)
{
   int i;
  
   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
__host__ int main(int argc, char *argv[])
{
    float *values,*final_result;

    int nsteps,                 	/* number of time steps */
        tpoints;    	     		/* total points along string */ 
    int blocknumber,datasize;
   
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
    blocknumber = tpoints/1024+1;
    datasize = (blocknumber * 1024 + 1) * sizeof(float);
	check_param(&tpoints,&nsteps);
    hipMalloc( (void**)&values,  datasize);
    final_result =(float *) malloc(datasize);
	
    printf("Initializing points on the line...\n");
    printf("Updating all points for all time steps...\n");
	init_line<<<blocknumber,1024>>>(values,tpoints,nsteps);


	printf("Printing final results...\n");
    hipMemcpy( final_result, values, datasize, hipMemcpyDeviceToHost ); 
	printfinal(final_result,tpoints);
	printf("\nDone.\n\n");
	hipFree(values);
	return 0;
}